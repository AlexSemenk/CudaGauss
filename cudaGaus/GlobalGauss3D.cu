#include "hip/hip_runtime.h"
#include "cuda_def.h"
#include <stdlib.h>
#include <stdio.h>
#include "matrix.h"

namespace global_gauss {

	#define BLOCK_X_SIZE 8		// r(2)
	#define BLOCK_Y_SIZE 8		// r(3)
	#define BLOCK_Z_SIZE 64		// r(1)

	#define R1 BLOCK_Z_SIZE
	#define R2 BLOCK_X_SIZE
	#define R3 BLOCK_Y_SIZE

	#define THREAD_X_SIZE 2 // r(2, 2)
	#define THREAD_Y_SIZE 2 // r(3, 2)
//	#define THERAK_K_SIZE - // r(1, 2)

	#define R22 THREAD_X_SIZE
	#define R32 THREAD_Y_SIZE
//	#define R12 THERAK_K_SIZE

	/*
	#TODO = SHARED MEMORY
	*/


	/*
	  ____________________________________________
	 |                                            |
	 |                                            |
	 |           _ _ _ _ _ _ _ _ _ _ _ _ _ _ _ _ _|
	 |          :_|____________const______________|
	 |          :c|_ _:_ _|       |       |       |
	 |          :o|___:___|_______|_______|_______|
	 |          :n|       |       |       |       |
	 |		    :s|_______|_______|_______|_______|
	 |          :t|       |       |       |       |
	 |__________:_|_______|_______|_______|_______|

	*/

	__global__ void solve_kernel(DeviceSystem<float> system, int k_gl, int t);
	__global__ void clean_kernel(DeviceSystem<float> system);
	void solve(DeviceSystem<float>& system);

	extern "C" void global_gauss_3d(System<float>& system) {
		size_t system_size = sizeof(float)*system.dim()*(system.dim()+1);
		DeviceSystem<float> dev_system((dev_size)system.dim());
		assertSuccess(hipSetDevice(0), "hipSetDevice(0) failed.");
		assertSuccess(hipMalloc(&dev_system.arr, system_size), "hipMalloc for device system failed.");
		assertSuccess(hipMemcpy(dev_system.arr, system.array, system_size, hipMemcpyHostToDevice), "hipMemcpy failed to copy system form hoste to device.");
		CUDA_TIME_OF(solve(dev_system));

		assertSuccess(hipMemcpy(system.array, dev_system.arr, system_size, hipMemcpyDeviceToHost), "hipMemcpy failed to copy system form device to host.");
		assertSuccess(hipFree(dev_system.arr), "hipFree failed to free device memory of system.");
		assertSuccess(hipDeviceReset(), "hipDeviceReset() deiled.");
	}

	void solve(DeviceSystem<float>& system) {
		int N = system.dim;
		int Q1 = div_ceiling(N-1, R1);
		// solve
		for (int k_gl=0; k_gl<Q1; k_gl++) {
			int Q2 = div_ceiling(N-1 - k_gl*R1, R2*THREAD_X_SIZE);
			int Q3 = div_ceiling(N - k_gl*R1, R3*THREAD_Y_SIZE);
			for (int t=1; t < Q2+Q3; t++) {
				int block_num;
				if (t < min(Q2, Q3)) {
					block_num = t;
				} else if (t > max(Q2, Q3)) {
					block_num = (Q2 + Q3) - t;
				} else {
					block_num = min(Q2, Q3);
				}
				dim3 blocks(block_num);
				dim3 threads(BLOCK_X_SIZE, BLOCK_Y_SIZE);
				//size_t sharedMemSize = sizeof(float) * BLOCK_X_SIZE * THREAD_X_SIZE * BLOCK_Y_SIZE * THREAD_Y_SIZE;
				solve_kernel<<<blocks, threads>>>(system, k_gl, t);
			}
		}
		// clean
		dim3 blocks(div_ceiling(N, 8), div_ceiling(N, 8));
		dim3 threads(8, 8);
		clean_kernel<<<blocks, threads>>>(system);
	}

	/* 

	     ri - relatiove i position
		 di - i position deviation

		 di = 0                       di = 1                       di = 4
		 dj = 0                       dj = 0                       dj = 1
		 ______________               ______________               ______________
		|____|####|____|  ri=0 rj=1  |____|____|____|             |____|____|____|
		|####|____|____|  ri=1 rj=0  |____|____|####|  ri=0 rj=2  |____|____|____|
		|____|____|____|             |____|####|____|  ri=1 rj=1  |____|____|____|
		|____|____|____|             |####|____|____|  ri=2 rj=0  |____|____|____|
		|____|____|____|             |____|____|____|             |____|____|####|  ri=0 rj=1
		|____|____|____|             |____|____|____|             |____|####|____|  ri=1 rj=0

	*/

	__global__ void solve_kernel(DeviceSystem<float> s, int k_gl, int t) {

		int N = s.dim;
		int start_k = k_gl*R1;
		int Q2 = dev_div_ceiling(N-1 - start_k, R2*THREAD_X_SIZE);
		int Q3 = dev_div_ceiling(N - start_k, R3*THREAD_Y_SIZE);
		
		int di_block = max(0, t - Q3);
		int dj_block = max(0, t - Q2);
		int ri_block = blockIdx.x;
		int rj_block = gridDim.x - 1 - blockIdx.x;
		int i_block = ri_block + di_block;
		int j_block = rj_block + dj_block;
	
		const int i_gl = (i_block * BLOCK_X_SIZE + threadIdx.x) * THREAD_X_SIZE + (start_k + 1);
		const int j_gl = (j_block * BLOCK_Y_SIZE + threadIdx.y) * THREAD_Y_SIZE + (start_k + 1);

		const int block_i_gl = i_block * BLOCK_X_SIZE * THREAD_X_SIZE + (start_k + 1);
		const int block_j_gl = j_block * BLOCK_Y_SIZE * THREAD_Y_SIZE + (start_k + 1);

		const int first_ai = threadIdx.x * THREAD_X_SIZE;
		const int first_aj = threadIdx.y * THREAD_Y_SIZE;

		const int sharedWidth = BLOCK_X_SIZE*THREAD_X_SIZE;
		const int sharedHeight = BLOCK_Y_SIZE*THREAD_Y_SIZE;
		__shared__ float a[sharedHeight][sharedWidth];
		for (int ai = first_ai, si = i_gl; si < min(i_gl + THREAD_X_SIZE, N); ai++, si++) {
			for(int aj = first_aj, sj = j_gl; sj < min(j_gl + THREAD_Y_SIZE, N+1); aj++, sj++) {
				a[ai][aj] = s[si][sj];
			}
		}

		for(int ks = start_k, kia = ks - block_i_gl, kja = ks - block_j_gl; ks < min(start_k+R1, N-1); ks++, kia++, kja++) {
			for (int is = max(i_gl, ks+1), ia = is - block_i_gl; is < min(i_gl + THREAD_X_SIZE, N); is++, ia++) {
				float aik = kja >= 0 && kja < sharedWidth ? a[ia][kja] : s[is][ks];
				float akk = kja >= 0 && kja < sharedWidth && kia >= 0 && kia < sharedHeight ? a[kia][kja] : s[ks][ks];
				float l = aik / akk;
				for(int js = max(j_gl, ks+1), ja = js - block_j_gl; js < min(j_gl + THREAD_Y_SIZE, N+1); js++, ja++) {
					float akj = ks < block_i_gl ? s[ks][js] : a[kia][ja];
					a[ia][ja] = a[ia][ja] - l * akj;
				}
			}
			__syncthreads();
		}

		for (int ai = first_ai, si = i_gl; si < min(i_gl + THREAD_X_SIZE, N); ai++, si++) {
			for(int aj = first_aj, sj = j_gl; sj < min(j_gl + THREAD_Y_SIZE, N+1); aj++, sj++) {
				s[si][sj] = a[ai][aj];
			}
		}

	}

	__global__ void clean_kernel(DeviceSystem<float> a) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		int j = blockIdx.y * blockDim.y + threadIdx.y;
		int N = a.dim;
		if (i<N && j<N && i>j) {
			a[i][j] = 0;
		}
	}

}